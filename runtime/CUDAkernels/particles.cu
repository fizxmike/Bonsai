#include "hip/hip_runtime.h"
#include "bonsai.h"
// #include "support_kernels.cu"
#include "../profiling/bonsai_timing.h"
PROF_MODULE(parallel);

#include <stdio.h>
#include "node_specs.h"

#include <cstdlib>
#include <iostream>
#include <map>
#include <cassert>
#include <algorithm>


#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/pair.h>
#include <thrust/partition.h>
#include <thrust/version.h>

#if THRUST_VERSION >=  100700


#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>

//Thrust cached allocator, note this assumes that the passed buffer has sufficient
//size for the requested operations!!!!!!!!!!!!!!
// cached_allocator: a simple allocator for caching allocation requests
class cached_allocator
{

private:
  int memOffset;
  int *currentPointer;

  public:

  typedef char value_type;

    cached_allocator(my_dev::dev_mem<uint> &temporaryBuffer, int startOffset)
    {
      memOffset      = 0;
      currentPointer = (int*)temporaryBuffer.a(startOffset);
    }

    ~cached_allocator(){}
    ///////////
    //Return the number of elements (of type uint) to be padded
    //to get to the correct address boundary
     int getGlobalMemAllignmentPadding2(int n)
    {
      const int allignBoundary = 128*sizeof(uint); //CC 2.X and 3.X ,128 bytes

      int offset = 0;
      //Compute the number of bytes
      offset = n*sizeof(uint);
      //Compute number of allignBoundary byte blocks
      offset = (offset / allignBoundary) + (((offset % allignBoundary) > 0) ? 1 : 0);
      //Compute the number of bytes padded / offset
      offset = (offset * allignBoundary) - n*sizeof(uint);
      //Back to the actual number of elements
      offset = offset / sizeof(uint);

      return offset;
    }


    char *allocate(std::ptrdiff_t num_bytes)
    {
      char *result = (char*)(void*)(size_t)(currentPointer + memOffset);

      //Convert num_bytes to integer offset
      int numIntItems = (int)(num_bytes / sizeof(int));
      numIntItems++;

      //Increase the offset, make sure it is a multiple of predefined number
      int currentOffset = memOffset + numIntItems;
      int padding       = getGlobalMemAllignmentPadding2(currentOffset);
      memOffset         = currentOffset + padding;

//      std::cout << "Allocating: bytes: " << num_bytes << std::endl;
//      std::cout << "Allocating: ints : " << numIntItems<< std::endl;
//      std::cout << "memOffset: " <<  memOffset  << std::endl;
//      std::cout << "currentOffset: " << currentOffset<< std::endl;
//      std::cout << "padding: " << padding << std::endl;

      return result;
    }

    void deallocate(char *ptr, size_t n) {}
};

#endif


struct isInOurDomain
{
  __host__ __device__
  bool operator()(const uint2 &val)
  {
    return (val.x >> 31);
  }
};

//struct isInOurDomain2
//{
//  __host__ __device__
//  bool operator()(const uint &val)
//  {
//    return (val >> 31);
//  }
//};

// user-defined comparison operator that acts like less<int>,
// except even numbers are considered to be smaller than odd numbers
struct domainCompare
{
  __host__ __device__
  bool operator()(uint2 x, uint2 y)
  {
    return x.x < y.x;
  }
};


 struct domainCompare2 : public binary_function<uint2,uint2,bool>
{
__host__ __device__ bool operator()(const uint2 &lhs, const uint2 &rhs) const {return lhs.x == rhs.x;}
}; // end domainCompare2

#include <sys/time.h>
 double get_time() {

   struct timeval Tvalue;
   struct timezone dummy;

   gettimeofday(&Tvalue,&dummy);
   return ((double) Tvalue.tv_sec +1.e-6*((double) Tvalue.tv_usec));
 }

extern "C" uint2 thrust_partitionDomains( my_dev::dev_mem<uint2> &validList,
                                          my_dev::dev_mem<uint2> &validList2, //Unsorted compacted list
                                          my_dev::dev_mem<uint> &idList,
                                          my_dev::dev_mem<uint2> &outputKeys,
                                          my_dev::dev_mem<uint> &outputValues,
                                          const int N,
                                          my_dev::dev_mem<uint> &generalBuffer,
                                          const int currentOffset)
{
  thrust::device_ptr<uint2> values      = thrust::device_pointer_cast(validList.raw_p());
  thrust::device_ptr<uint>  listofones  = thrust::device_pointer_cast(idList.raw_p());
  thrust::device_ptr<uint2> outKeys     = thrust::device_pointer_cast(outputKeys.raw_p());
  thrust::device_ptr<uint>  outValues   = thrust::device_pointer_cast(outputValues.raw_p());
  thrust::plus<int> binary_op;


#if THRUST_VERSION >=  100700
  cached_allocator alloc(generalBuffer, currentOffset);

  double t1 = get_time();
  //Partition the values by in or out of domain. Result: [[outside],[inside ids]]
  thrust::device_ptr<uint2>  res = thrust::partition(thrust::cuda::par(alloc), values, values + N, isInOurDomain());
  const int remoteParticles      = (int) (res-values);
  double t2 = get_time();

  validList2.copy_devonly(validList, remoteParticles); //Copy the list before sorting, needed for internal move

  //Sort the outside our domain particles by their domain index
  //Result: [[ids domain0],[ids domain1], [ids domain2], ...]
#if 0
  //Although it is faster it puts particles in the wrong order because we have marked
  //them with a high bit integer. Also does not work to put domains in right order. So
  //not use this for now. Can look at it at a later point
  unsigned long long *tempPtr = (unsigned long long*)validList.raw_p();
  thrust::device_ptr<unsigned long long> valuesLL      = thrust::device_pointer_cast(tempPtr);
  thrust::stable_sort(thrust::cuda::par(alloc),
                      valuesLL,
                      valuesLL + remoteParticles,
                      thrust::greater<unsigned long long>());
#else
  thrust::stable_sort(thrust::cuda::par(alloc),
                      values,
                      values + remoteParticles,
                      domainCompare());
//  hipDeviceSynchronize();
#endif
  double t3 = get_time();
  //Reduce the domains. The result is that we get per domain the number of particles
  //that will send to that process. These are stored into the output buffers
  thrust::pair<thrust::device_ptr<uint2>,thrust::device_ptr<uint> > new_end;
  new_end = thrust::reduce_by_key(thrust::cuda::par(alloc),
                                  values,                   //inputIterator1
                                  values + remoteParticles, //InputIterator1
                                  listofones,              //InputIterator2
                                  outKeys,                  //OutputIterator1
                                  outValues,                  //OutputIterator2
                                  domainCompare2(),
                                  binary_op);

#else
  //Partition the values by in or out of domain. Result: [[outside],[inside ids]]
   double t1 = get_time();
  thrust::device_ptr<uint2>  res = thrust::partition(values, values + N, isInOurDomain());
  const int remoteParticles      = (int) (res-values);
  double t2 = get_time();
  validList2.copy_devonly(validList, remoteParticles); //Copy the list before sorting, needed for internal move

  //Sort the outside our domain particles by their domain index
  //Result: [[ids domain0],[ids domain1], [ids domain2], ...]
  thrust::stable_sort(values,  values + remoteParticles, domainCompare());
  //hipDeviceSynchronize();
  double t3 = get_time();
  //Reduce the domains. The result is that we get per domain the number of particles
  //that will send to that process. These are stored into the output buffers
  thrust::pair<thrust::device_ptr<uint2>,thrust::device_ptr<uint> > new_end;
  new_end = thrust::reduce_by_key(values,                   //inputIterator1
                                  values + remoteParticles, //InputIterator1
                                  listofones,              //InputIterator2
                                  outKeys,                  //OutputIterator1
                                  outValues,                  //OutputIterator2
                                  domainCompare2(),
                                  binary_op);
#endif

  LOGF(stderr,"Sorting detail: N: %d partition: %lg sort: %lg reduce: %lg \n",remoteParticles, t2-t1,t3-t2,get_time()-t3);

  const int nValues = (int)(new_end.first  - outKeys);
 //return the number of remote particles and the number of remote domains
 return make_uint2(remoteParticles, nValues);

}











static __device__ inline int isinbox(real4 pos, double4 xlow, double4 xhigh)
{  
    if((pos.x < xlow.x)||(pos.x > xhigh.x))          
      return 0;
    if((pos.y < xlow.y)||(pos.y > xhigh.y))          
      return 0;
    if((pos.z < xlow.z)||(pos.z > xhigh.z))          
      return 0;
    
    return 1;
}

static __device__ inline int cmp_uint4(uint4 a, uint4 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    else {
      if       (a.z < b.z) return -1;
      else  if (a.z > b.z) return +1;
      return 0;
    } //end z
  }  //end y
} //end x, function



KERNEL_DECLARE(doDomainCheck)(int    n_bodies,
                                           double4  xlow,
                                           double4  xhigh,
                                           real4  *body_pos,
                                           int    *validList    //Valid is 1 if particle is outside domain
){
  CUXTIMER("doDomainCheck");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id >= n_bodies) return;

  real4 pos = body_pos[id];

  int valid      = isinbox(pos, xlow, xhigh);
  valid = !valid;
  validList[id] = id | ((valid) << 31);
}
  

//Checks the domain and computes the key list
//if a particle is outside the domain it gets a special key
//otherwise the normal key is used
KERNEL_DECLARE(doDomainCheckAdvanced)(int    n_bodies,
                                           double4  xlow,
                                           double4  xhigh,
                                           real4  *body_pos,
                                           int    *validList    //Valid is 1 if particle is outside domain
){
  CUXTIMER("doDomainCheckAdvanced");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id >= n_bodies) return;

  real4 pos = body_pos[id];

  int valid      = isinbox(pos, xlow, xhigh);
  valid = !valid;
  validList[id] = id | ((valid) << 31);
}
  

KERNEL_DECLARE(gpu_extractSampleParticles)(int    n_bodies,
                                                  int    sample_freq,
                                                  real4  *body_pos,
                                                  real4  *samplePosition
){
  CUXTIMER("extractSampleParticles");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  

  int idx  = id*sample_freq;
  if  (idx >= n_bodies) return;

  samplePosition[id] =  body_pos[idx];
}

KERNEL_DECLARE(gpu_extractSampleParticlesSFC)(int     n_bodies,
                                              int     nSamples,
                                              float   sample_freq,
                                              uint4  *body_pos,
                                              uint4  *samplePosition
){
  CUXTIMER("extractSampleParticles");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= nSamples) return;

  int idx  = (int)(id*sample_freq);
  if  (idx >= n_bodies) return;

  samplePosition[id] =  body_pos[idx];
}

KERNEL_DECLARE(extractOutOfDomainParticlesR4)(int n_extract,
                                                       int *extractList,
                                                       real4 *source,
                                                       real4 *destination)
{
  CUXTIMER("extractOutOfDomainParticlesR4");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  destination[id] = source[extractList[id]];

}



typedef struct bodyStruct
{
  real4 pos;
  real4 vel;
  real4 acc0;
  real4 acc1;
  real4 Ppos;
  real4 Pvel;
  float2 time;
  int   id;
  int   temp;
  uint4 key;
} bodyStruct;


KERNEL_DECLARE(extractOutOfDomainParticlesAdvanced)(int n_extract,
                                                       int *extractList,
                                                       real4 *Ppos,
                                                       real4 *Pvel,
                                                       real4 *pos,
                                                       real4 *vel,
                                                       real4 *acc0,
                                                       real4 *acc1,
                                                       float2 *time,
                                                       int   *body_id,
                                                       bodyStruct *destination)
{
  CUXTIMER("extractOutOfDomainParticlesAdvanced");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  //copy the data from a struct of arrays into a array of structs
  destination[id].Ppos = Ppos[extractList[id]];
  destination[id].Pvel = Pvel[extractList[id]];
  destination[id].pos  = pos[extractList[id]];
  destination[id].vel  = vel[extractList[id]];
  destination[id].acc0  = acc0[extractList[id]];
  destination[id].acc1  = acc1[extractList[id]];
  destination[id].time  = time[extractList[id]];
  destination[id].id    = body_id[extractList[id]];

}


KERNEL_DECLARE(gpu_internalMove)(int       n_extract,
                                        int       n_bodies,
                                        double4  xlow,
                                        double4  xhigh,
                                        int       *extractList,
                                        int       *indexList,
                                        real4     *Ppos,
                                        real4     *Pvel,
                                        real4     *pos,
                                        real4     *vel,
                                        real4     *acc0,
                                        real4     *acc1,
                                        float2    *time,
                                        int       *body_id)
{
  CUXTIMER("internalMove");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;
  real4 testpos  = Ppos[srcIdx];

  if(isinbox(testpos, xlow, xhigh))
  {
    int dstIdx = atomicAdd(indexList, 1);    
    dstIdx     = extractList[dstIdx];

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_id[dstIdx] = body_id[srcIdx];
  }//if isinbox

}

KERNEL_DECLARE(gpu_insertNewParticles)(int       n_extract,
                                              int       n_insert,
                                              int       n_oldbodies,
                                              int       offset,
                                              real4     *Ppos,
                                              real4     *Pvel,
                                              real4     *pos,
                                              real4     *vel,
                                              real4     *acc0,
                                              real4     *acc1,
                                              float2    *time,
                                              int       *body_id,
                                              bodyStruct *source)
{
  CUXTIMER("insertNewParticles");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_insert) return;

  //The newly added particles are added at the end of the array
  int idx = (n_oldbodies-n_extract) + id + offset;

  //copy the data from a struct of arrays into a array of structs
  Ppos[idx]     = source[id].Ppos;
  Pvel[idx]     = source[id].Pvel;
  pos[idx]      = source[id].pos;
  vel[idx]      = source[id].vel;
  acc0[idx]     = source[id].acc0;
  acc1[idx]     = source[id].acc1;
  time[idx]     = source[id].time;
  body_id[idx]  = source[id].id;
}

//Check if a particles key is within the min and max boundaries
KERNEL_DECLARE(gpu_domainCheckSFC)(int    n_bodies,
                               uint4  lowBoundary,
                               uint4  highBoundary,
                               uint4  *body_key,
                               int    *validList    //Valid is 1 if particle is outside domain
){
  CUXTIMER("domainCheckSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if (id >= n_bodies) return;

  uint4 key = body_key[id];

  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);

  int valid = 0;
  if(bottom >= 0 && top < 0)
  {
    //INside
  }
  else
  {
    //    outside
    valid = 1;
  }
  validList[id] = id | ((valid) << 31);
}


//Binary search of the key within certain bounds (cij.x, cij.y)
//Note this is the same as 'find_key'
static __device__ int find_domain(uint4 key, uint2 cij, uint4 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = cmp_uint4(keys[m], key);
    if (cmp == -1) {
      l = m;
    } else {
      r = m;
    }
  }
  if (cmp_uint4(keys[l], key) >= 0) return l;

  return r;
}
//Check if a particles key is within the min and max boundaries
KERNEL_DECLARE(gpu_domainCheckSFCAndAssign)(int    n_bodies,
                                            int    nProcs,
                                            uint4  lowBoundary,
                                            uint4  highBoundary,
                                            uint4  *boundaryList, //The full list of boundaries
                                            uint4  *body_key,
                                            uint2  *validList,    //Valid is 1 if particle is outside domain,
                                            uint   *idList
){
  CUXTIMER("domainCheckSFCAndAssign");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if (id >= n_bodies) return;

  uint4 key = body_key[id];

  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);

  uint valid = 0;
  if(bottom >= 0 && top < 0)
  {
    //Inside
//    valid = 0x0;
  }
  else
  {
    //    outside
    //Search the box that this particle belongs to. Note we start at idx[1] that
    //way we get the top-end values of the domain
    uint2 cij;
    cij.x = 0; cij.y = nProcs+1;
    int domain = find_domain(key, cij, &boundaryList[1]);

//    valid = domain;
    valid = domain | ((1) << 31);
  }
  //validList[id] = id | ((valid) << 31);
  validList[id] = make_uint2(valid, id);
  idList[id]    = 1;
}

#if 0
//Check if a particles key is within the min and max boundaries
KERNEL_DECLARE(gpu_domainCheckSFCAndAssign)(int    n_bodies,
                                            int    nProcs,
                                            uint4  lowBoundary,
                                            uint4  highBoundary,
                                            uint4  *boundaryList, //The full list of boundaries
                                            uint4  *body_key,
                                            int    *validList    //Valid is 1 if particle is outside domain
){
  CUXTIMER("domainCheckSFCAndAssign");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if (id >= n_bodies) return;

  uint4 key = body_key[id];

  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);

  int valid = 0;
  if(bottom >= 0 && top < 0)
  {
    //Inside
    valid = -1;
  }
  else
  {
    //    outside
    valid = 1;

//    if(id == 4992)
    {
      //Search the box that this particle belongs to
      uint2 cij;
      cij.x = 0; cij.y = nProcs+1;
      int domain = find_domainBox(key, cij, boundaryList);


//      printf("XXX Particles: %d outside, namely in domain: %d \n", id, domain);
      valid = domain;
    }


  }
  //validList[id] = id | ((valid) << 31);
  validList[id] = valid;
}
#endif

KERNEL_DECLARE(gpu_internalMoveSFC2) (int       n_extract,
                                  int       n_bodies,
                                  uint4  lowBoundary,
                                  uint4  highBoundary,
                                  int2       *extractList,
                                  int       *indexList,
                                  real4     *Ppos,
                                  real4     *Pvel,
                                  real4     *pos,
                                  real4     *vel,
                                  real4     *acc0,
                                  real4     *acc1,
                                  float2    *time,
                                  int       *body_id,
                                  uint4     *body_key)
{
  CUXTIMER("internalMoveSFC2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;


  uint4 key  = body_key[srcIdx];
  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);


  if((bottom >= 0 && top < 0))
  {
    int dstIdx = atomicAdd(indexList, 1);
    dstIdx     = extractList[dstIdx].y;

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_key[dstIdx] = body_key[srcIdx];
    body_id[dstIdx]  = body_id[srcIdx];
  }//if inside

}

KERNEL_DECLARE(gpu_internalMoveSFC) (int       n_extract,
                                  int       n_bodies,
                                  uint4  lowBoundary,
                                  uint4  highBoundary,
                                  int       *extractList,
                                  int       *indexList,
                                  real4     *Ppos,
                                  real4     *Pvel,
                                  real4     *pos,
                                  real4     *vel,
                                  real4     *acc0,
                                  real4     *acc1,
                                  float2    *time,
                                  int       *body_id,
                                  uint4     *body_key)
{
  CUXTIMER("internalMoveSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;


  uint4 key  = body_key[srcIdx];
  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);


  if((bottom >= 0 && top < 0))
  {
    int dstIdx = atomicAdd(indexList, 1);
    dstIdx     = extractList[dstIdx];

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_key[dstIdx] = body_key[srcIdx];
    body_id[dstIdx]  = body_id[srcIdx];
  }//if inside

}

KERNEL_DECLARE(gpu_extractOutOfDomainParticlesAdvancedSFC2)(
                                                       int offset,
                                                       int n_extract,
                                                       uint2 *extractList,
                                                       real4 *Ppos,
                                                       real4 *Pvel,
                                                       real4 *pos,
                                                       real4 *vel,
                                                       real4 *acc0,
                                                       real4 *acc1,
                                                       float2 *time,
                                                       int   *body_id,
                                                       uint4 *body_key,
                                                       bodyStruct *destination)
{
  CUXTIMER("extractOutOfDomainParticlesAdvancedSFC2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
#if 0
  //slowest
  if(id >= n_extract) return;

  //copy the data from a struct of arrays into a array of structs
  destination[id].Ppos = Ppos[extractList[offset+id].y];
  destination[id].Pvel = Pvel[extractList[offset+id].y];
  destination[id].pos  = pos[extractList[offset+id].y];
  destination[id].vel  = vel[extractList[offset+id].y];
  destination[id].acc0  = acc0[extractList[offset+id].y];
  destination[id].acc1  = acc1[extractList[offset+id].y];
  destination[id].time  = time[extractList[offset+id].y];
  destination[id].id    = body_id[extractList[offset+id].y];
  destination[id].key   = body_key[extractList[offset+id].y];

#elif 1
  //5x faster than original (above)
  __shared__ bodyStruct shmem[128];

  if((bid * blockDim.x) >= n_extract) return;

  if(id < n_extract)
  {
    shmem[threadIdx.x].Ppos  = Ppos[extractList[offset+id].y];
    shmem[threadIdx.x].Pvel  = Pvel[extractList[offset+id].y];
    shmem[threadIdx.x].pos   = pos[extractList[offset+id].y];
    shmem[threadIdx.x].vel   = vel[extractList[offset+id].y];
    shmem[threadIdx.x].acc0  = acc0[extractList[offset+id].y];
    shmem[threadIdx.x].acc1  = acc1[extractList[offset+id].y];
    shmem[threadIdx.x].time  = time[extractList[offset+id].y];
    shmem[threadIdx.x].id    = body_id[extractList[offset+id].y];
    shmem[threadIdx.x].key   = body_key[extractList[offset+id].y];
  }
  __syncthreads();

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];


  //We have blockDim.x thread, each thread writes a float4. Compute number of items per thread-block
  //and number of loops and remaining items
  const float nThreadsPerItem = sizeof(bodyStruct) / sizeof(float4);
  const int   nItemsPerLoop   = (int)(blockDim.x / nThreadsPerItem);

  const int nExtractThisBlock = min(n_extract-startWrite, (int)blockDim.x);

  const int   nLoops          = (nExtractThisBlock/nItemsPerLoop);
  int         nExtra          = (nExtractThisBlock - nLoops*nItemsPerLoop)*nThreadsPerItem;


  int startOut = 0;
  for(int i=0; i < nLoops; i++)
  {
    output[startOut + threadIdx.x] = shdata4[threadIdx.x  + startOut]; //Write blockDim.x * float4 items
    startOut += blockDim.x;
  }

  //Write the remaining items
  if(threadIdx.x < nExtra)
  {
    output[startOut + threadIdx.x] = shdata4[threadIdx.x  + startOut]; //Write remaining float4 items
  }



#endif

#if 0
  Do not use the below kernels without first checking
  that the offsets are correct. There were problems with the ones
  above.

#elif 0
  //
  __shared__ bodyStruct shmem[64];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startOut = 0;
  for(int j=0; j < 2; j++)
  {
    int nExtractThisBlock = min(n_extract-(bid * blockDim.x), (int)blockDim.x);
    nExtractThisBlock    -= j*64;
    nExtractThisBlock     = min(64, nExtractThisBlock);

    int readIdx = j*64+offset+id;

    __syncthreads();
    if(warpId < 2)
    {
      if(readIdx < n_extract)
      {
        readIdx                  = extractList[readIdx].y;
        shmem[threadIdx.x].Ppos  = Ppos[readIdx];
        shmem[threadIdx.x].Pvel  = Pvel[readIdx];
        shmem[threadIdx.x].pos   = pos[readIdx];
        shmem[threadIdx.x].vel   = vel[readIdx];
      }
    }
    else
    {
      if(readIdx-64 < n_extract)
      {
        readIdx                       = extractList[readIdx- 64].y;
        shmem[threadIdx.x - 64].acc0  = acc0[readIdx];
        shmem[threadIdx.x - 64].acc1  = acc1[readIdx];
        shmem[threadIdx.x - 64].time  = time[readIdx];
        shmem[threadIdx.x - 64].id    = body_id[readIdx];
        shmem[threadIdx.x - 64].key   = body_key[readIdx];
      }
    }
    __syncthreads();


    int startWrite  = bid * blockDim.x;
    float4 *shdata4 = (float4*)shmem;
    float4 *output  = (float4*)&destination[startWrite];

    //We have blockDim.x thread, each thread writes a float4. Compute number of items per thread-block
    //and number of loops and remaining items
    const float nThreadsPerItem = sizeof(bodyStruct) / sizeof(float4);
    const int   nItemsPerLoop   = (int)(blockDim.x / nThreadsPerItem);

    const int   nLoops          = (nExtractThisBlock/nItemsPerLoop);
    const int   nExtra          = (nExtractThisBlock - nLoops*nItemsPerLoop)*nThreadsPerItem;


  #pragma unroll
    for(int i=0; i < nLoops; i++)
    {
      output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

    //Write the remaining items
    if(threadIdx.x < nExtra)
    {
      output[startOut + threadIdx.x] = shdata4[threadIdx.x  + nLoops*blockDim.x]; //Write first blockDim.x * float4 items
    }
  } //for j

#elif 0

//Comparable to one below

  __shared__ bodyStruct shmem[32];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];
  int startOut    = 0;

  float4 temp1;
  float4 temp2;
  uint4  temp3;
  float2 temp4;
  int    temp5;

  for(int loop=0; loop < 4; loop++) //4 = 128 threads / 32 items
  {
    int readIdx = (bid * blockDim.x) + loop*32 + laneId + offset;
    //We only read as much as we can write in two transaction
    readIdx = min(readIdx, n_extract-1);

    if(warpId == 0)
    {
      temp1 = pos[extractList[readIdx].y];
      temp2 = Ppos[extractList[readIdx].y];
      temp3 = body_key[extractList[readIdx].y];
    }
    else if (warpId == 1)
    {
      temp1 = Pvel[extractList[readIdx].y];
      temp2 = vel[extractList[readIdx].y];
    }
    else if (warpId == 2)
    {
      temp1 = acc0[extractList[readIdx].y];
      temp2 = acc1[extractList[readIdx].y];
    }
    else if (warpId == 3)
    {
      temp5   = body_id[extractList[readIdx].y];
      temp4 = time[extractList[readIdx].y];
    }

    __syncthreads();

    if(warpId == 0)
    {
      shmem[laneId].pos = temp1;
      shmem[laneId].Ppos = temp2;
      shmem[laneId].key = temp3;
    }
    else if (warpId == 1)
    {
      shmem[laneId].Pvel = temp1;
      shmem[laneId].vel = temp2;
    }
    else if (warpId == 2)
    {
      shmem[laneId].acc0 = temp1;
      shmem[laneId].acc1 = temp2;
    }
    else if (warpId == 3)
    {
      shmem[laneId].id   = temp5;
      shmem[laneId].time = temp4;
    }
    __syncthreads();

    for(int i=0; i < 2; i++) //2 is 32 items / 16 items write per 128 threads
    {
      if(startOut + threadIdx.x < (8*n_extract)) //8* sincce 8 float4 in bodystruct
        output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

  }
#elif 0
//Second fastest
  __shared__ bodyStruct shmem[32];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];
  int startOut    = 0;

  for(int loop=0; loop < 4; loop++) //4 = 128 threads / 32 items
  {
    int readIdx = (bid * blockDim.x) + loop*32 + laneId  + offset;
    //We only read as much as we can write in two transaction
    readIdx = min(readIdx, n_extract-1);

    __syncthreads();
    if(warpId == 0)
    {
      shmem[laneId].pos = pos[extractList[readIdx].y];
      shmem[laneId].Ppos = Ppos[extractList[readIdx].y];
      shmem[laneId].key = body_key[extractList[readIdx].y];
    }
    else if (warpId == 1)
    {
      shmem[laneId].Pvel = Pvel[extractList[readIdx].y];
      shmem[laneId].vel = vel[extractList[readIdx].y];
    }
    else if (warpId == 2)
    {
      shmem[laneId].acc0 = acc0[extractList[readIdx].y];
      shmem[laneId].acc1 = acc1[extractList[readIdx].y];
    }
    else if (warpId == 3)
    {
      shmem[laneId].id   = body_id[extractList[readIdx].y];
      shmem[laneId].time = time[extractList[readIdx].y];
    }

    __syncthreads();

    for(int i=0; i < 2; i++) //2 is 32 items / 16 items write per 128 threads
    {
      if(startOut + threadIdx.x < (8*n_extract)) //8* sincce 8 float4 in bodystruct
        output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

  }

#endif


}

KERNEL_DECLARE(gpu_extractOutOfDomainParticlesAdvancedSFC)(
                                                       int offset,
                                                       int n_extract,
                                                       int *extractList,
                                                       real4 *Ppos,
                                                       real4 *Pvel,
                                                       real4 *pos,
                                                       real4 *vel,
                                                       real4 *acc0,
                                                       real4 *acc1,
                                                       float2 *time,
                                                       int   *body_id,
                                                       uint4 *body_key,
                                                       bodyStruct *destination)
{
  CUXTIMER("extractOutOfDomainParticlesAdvancedSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  //copy the data from a struct of arrays into a array of structs
  destination[id].Ppos = Ppos[extractList[offset+id]];
  destination[id].Pvel = Pvel[extractList[offset+id]];
  destination[id].pos  = pos[extractList[offset+id]];
  destination[id].vel  = vel[extractList[offset+id]];
  destination[id].acc0  = acc0[extractList[offset+id]];
  destination[id].acc1  = acc1[extractList[offset+id]];
  destination[id].time  = time[extractList[offset+id]];
  destination[id].id    = body_id[extractList[offset+id]];
  destination[id].key   = body_key[extractList[offset+id]];
}

KERNEL_DECLARE(gpu_insertNewParticlesSFC)(int       n_extract,
                                              int       n_insert,
                                              int       n_oldbodies,
                                              int       offset,
                                              real4     *Ppos,
                                              real4     *Pvel,
                                              real4     *pos,
                                              real4     *vel,
                                              real4     *acc0,
                                              real4     *acc1,
                                              float2    *time,
                                              int       *body_id,
                                              uint4     *body_key,
                                              bodyStruct *source)
{
  CUXTIMER("insertNewParticlesSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_insert) return;

  //The newly added particles are added at the end of the array
  int idx = (n_oldbodies-n_extract) + id + offset;

  //copy the data from a struct of arrays into a array of structs
  Ppos[idx]     = source[id].Ppos;
  Pvel[idx]     = source[id].Pvel;
  pos[idx]      = source[id].pos;
  vel[idx]      = source[id].vel;
  acc0[idx]     = source[id].acc0;
  acc1[idx]     = source[id].acc1;
  time[idx]     = source[id].time;
  body_id[idx]  = source[id].id;
  body_key[idx] = source[id].key;
}



// KERNEL_DECLARE(insertNewParticles)(int       n_extract,
//                                               int       n_insert,
//                                               int       n_oldbodies,
//                                               int       *extractList,
//                                               real4     *Ppos,
//                                               real4     *Pvel,
//                                               real4     *pos,
//                                               real4     *vel,
//                                               real4     *acc0,
//                                               real4     *acc1,
//                                               float2    *time,
//                                               int       *body_id,
//                                               bodyStruct *source)
// {
//   uint bid = blockIdx.y * gridDim.x + blockIdx.x;
//   uint tid = threadIdx.x;
//   uint id  = bid * blockDim.x + tid;
// 
//   int idx, srcidx = -1; 
// /*
// 
// //Situaties:
// - n_insert > n_extract -> particles moeten aan einde worden toegevoegd (meer toevoegen dan weggehaald)
//     id < n_extract -> idx = extractList[id]  ; uit source[id]
//     id >= n_extract & id < n_insert  --> idx = n_oldbodies + (id-n_extract); uit source[id]
//   
// - n_insert <= n_exract -> particles moeten van het einde naar het begin (meer verwijderd dan toegevoegd)
//     id < n_extract -> idx = extractList[id] ; uit source[id]
//     id >= n_extract & id < n_insert -> idx = extractList[id] ; uit dest[n_bodies-(n_extract-n_insert) + (id - n_insert)]
// 
//   */
// 
//   if(n_insert > n_extract)
//   {
//     if(id < n_extract)
//     {
//        idx = extractList[id];
//     }
//     else if(id >= n_extract && id < n_insert)
//     {
//       //Insert particles at the end of the array
//       idx = n_oldbodies + (id-n_extract);
//     }
//     else
//     {
//       return;
//     }
//   }
//   else
//   {
//     //n_insert <= n_extract
// 
//     if(id < n_insert)
//     {
//        idx = extractList[id];
//     }
//     else if(id >= n_insert && id < n_extract)
//     {
//       //Move particles from the back of the array to the empty spots
//       idx    = extractList[id];
//       srcidx = extractList[n_oldbodies-(n_extract-n_insert) + (id - n_insert)];
//     //  srcidx = n_oldbodies-(n_extract-n_insert) + (id - n_insert);
//     }
//     else
//     {
//       return;
//     }
//   }
// /*
// Gaat niet goed als n_insert < n_extract
// omdat we als we gaan moven we ook kans hebben dat we iets moven
// van het begin naar het eind als daar iets is uitgehaald
// we zouden dus de laatste verwijderde moeten vinden en zorgen dat er neits achter komt ofzo
// 
// 
// 
// */
// 
// /*
//   if(id < n_extract)
//   {
//     idx = extractList[id];
//   }
//   else if(id >= n_extract && id < n_insert)
//   {
//     if(n_insert > n_extract)
//     {
//       //Insert particles at the end of the array
//       idx = n_oldbodies + (id-n_extract);
//     }
//     else
//     {
//       //Move particles from the back of the array to the empty spots
//       idx    = extractList[id];
//       srcidx = n_oldbodies-(n_extract-n_insert) + (id - n_insert);
//     }
//   }
//   else
//   {
//     //Outside all array ranges
//     return;
//   }*/
// 
// 
//   if(srcidx < 0)
//   {
//     //copy the data from a struct of arrays into a array of structs
//     Ppos[idx] = source[id].Ppos;
//     Pvel[idx] = source[id].Pvel;
//     pos[idx]  = source[id].pos;
//     vel[idx]  = source[id].vel;
//     acc0[idx] = source[id].acc0;
//     acc1[idx] = source[id].acc1;
//     time[idx] = source[id].time;
//     body_id[idx] = source[id].id;
// 
// printf("%d  (CMOVE external %d) goes to: %d \n", source[id].id,n_insert, idx);
// 
// 
//   }
//   else
//   {
//     Ppos[idx] = Ppos[srcidx];
//     Pvel[idx] = Pvel[srcidx];
//     pos[idx]  = pos[srcidx];
//     vel[idx]  = vel[srcidx];
//     acc0[idx] = acc0[srcidx];
//     acc1[idx] = acc1[srcidx];
//     time[idx] = time[srcidx];
//   int temp = body_id[idx];
//     body_id[idx] = body_id[srcidx];
// 
// printf("%d stored at: %d (CMOVE internal %d) goes to: %d  overwr: %d \n", body_id[srcidx],srcidx, n_insert, idx, temp);
// 
// 
//   }//if srcidx < 0
// 
// 
// }


