#include "hip/hip_runtime.h"
#include "bonsai.h"

#include "support_kernels.cu"
#include <stdio.h>

#include "../profiling/bonsai_timing.h"
PROF_MODULE(build_tree);


KERNEL_DECLARE(gpu_boundaryReduction)(const int n_particles,
                                            real4      *positions,
                                            float3     *output_min,
                                            float3     *output_max)
{
  CUXTIMER("boundaryReduction");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_particles) {
    if (i             < n_particles)
    {
      pos = positions[i];
      r_min.x = fminf(pos.x, r_min.x);
      r_min.y = fminf(pos.y, r_min.y);
      r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x);
      r_max.y = fmaxf(pos.y, r_max.y);
      r_max.z = fmaxf(pos.z, r_max.z);
    }
    if (i + blockSize < n_particles)
    {
      pos = positions[i + blockSize];
      r_min.x = fminf(pos.x, r_min.x);
      r_min.y = fminf(pos.y, r_min.y);
      r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x);
      r_max.y = fmaxf(pos.y, r_max.y);
      r_max.z = fmaxf(pos.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
  }

}

//Get the domain size, by taking into account the group size
KERNEL_DECLARE(gpu_boundaryReductionGroups)(const int n_groups,
                                                   real4      *positions,
                                                   real4      *sizes,
                                                   float3     *output_min,
                                                   float3     *output_max)
{
  CUXTIMER("boundaryReductionGroups");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  real4 size;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_groups) {
    if (i             < n_groups)
    {
      pos = positions[i];
      size = sizes[i];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    if (i + blockSize < n_groups)
    {
      pos = positions[i + blockSize];
      size = sizes[i + blockSize];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
  }

}

//#define EXACT_KEY

KERNEL_DECLARE(cl_build_key_list)(uint4  *body_key,
                                            real4  *body_pos,
                                            int   n_bodies,
                                            real4  corner) {
  
  CUXTIMER("cl_build_key_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id > n_bodies) return;

  real4 pos = body_pos[id];

  int4 crd;
  
  real domain_fac = corner.w;
  
  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif

  uint4 key = get_key(crd);

  if (id == n_bodies) key = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0, 0);

  key.w        = id;
  body_key[id] = key;
}
  

KERNEL_DECLARE(cl_build_valid_list)(int n_bodies,
                                               int level,
                                               uint4  *body_key,
                                               uint *valid_list,
                                               const uint *workToDo) {
  if (0 == *workToDo) return;
//                                                uint2 *test_key_data) {
  CUXTIMER("cl_build_valid_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  const uint4 key_F = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
  
  if (id >= n_bodies) return;   // >=   since the last particle is extra boudnary particle
  
  uint4 mask = get_mask(level);
  mask.x = mask.x | ((uint)1 << 30) | ((uint)1 << 31);

  uint4 key_m;
  uint4 key_c    = body_key[id];
  uint4 key_p;

  if (id == 0)
  {
    key_m = key_F;
  }
  else
  {
    key_m = body_key[id-1];
  }

  if((id+1) <  n_bodies) //The last particle gets a different key to compare with
  {
    key_p = body_key[id+1];
  }
  else
    key_p = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF);


  int valid0 = 0;
  int valid1 = 0;

  if (cmp_uint4(key_c, key_F) != 0) {
    key_c.x = key_c.x & mask.x;
    key_c.y = key_c.y & mask.y;
    key_c.z = key_c.z & mask.z;

    key_p.x = key_p.x & mask.x;
    key_p.y = key_p.y & mask.y;
    key_p.z = key_p.z & mask.z;

    key_m.x = key_m.x & mask.x;
    key_m.y = key_m.y & mask.y;
    key_m.z = key_m.z & mask.z;

    valid0 = abs(cmp_uint4(key_c, key_m));
    valid1 = abs(cmp_uint4(key_c, key_p));
  }

   valid_list[id*2]   = id | ((valid0) << 31);
   valid_list[id*2+1] = id | ((valid1) << 31);

}


//////////////////////////////
//////////////////////////////
//////////////////////////////
static __device__ uint retirementCountBuildNodes = 0;

KERNEL_DECLARE(cl_build_nodes)(uint level,
                             uint  *compact_list_len,
                             uint  *level_offset,
                             uint  *last_level,
                             uint2 *level_list,
                             uint  *compact_list,
                             uint4 *bodies_key,
                             uint4 *node_key,
                             uint  *n_children,
                             uint2 *node_bodies){

  CUXTIMER("cl_build_nodes");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  uint n = (*compact_list_len)/2;
  uint offset = *level_offset;

  //We reuse last_level as indicator if we are allowed to create LEAF nodes
  bool minLevelReached = (int)*last_level;

  for (; id < n; id += gridDim.x * gridDim.y * blockDim.x)
  {
    uint  bi   = compact_list[id*2];
    uint  bj   = compact_list[id*2+1] + 1;
  
    uint4 key  = bodies_key[bi];
    uint4 mask = get_mask(level);
    key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

    node_bodies[offset+id] = make_uint2(bi | (level << BITLEVELS), bj);
    node_key   [offset+id] = key;
    n_children [offset+id] = 0;
  
//    if ((int)level > (int)(LEVEL_MIN - 1))
    if(minLevelReached)
      if (bj - bi <= NLEAF)                            //Leaf can only have NLEAF particles, if its more there will be a split
        for (int i = bi; i < bj; i++)
          bodies_key[i] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF); //sets the key to FF to indicate the body is used
  }

  //
  // PHASE 2: Last block updates level list and offset
  //

  int numBlocks = gridDim.x * gridDim.y;
  if (numBlocks > 1)
  {
    __shared__ bool amLast;

    // Thread 0 takes a ticket
    if( tid==0 )
    {
      unsigned int ticket = atomicInc(&retirementCountBuildNodes, numBlocks);
      // If the ticket ID is equal to the number of blocks, we are the last block!
      amLast = (ticket == numBlocks-1);
    }
    __syncthreads();

    // The last block sums the results of all other blocks
    if( amLast && tid == 0)
    {           
      level_list[level] = (n > 0) ? make_uint2(offset, offset + n) : make_uint2(0, 0);
      *level_offset = offset + n;


      //Set last_level to a value to indicate we are now allowed to make
      //leafs. It will later be overwritten to indicate the final level
      if(n > START_LEVEL_MIN_NODES){
          *last_level = 1;
      }

      if ((level > 0) && (n <= 0) && (level_list[level - 1].x > 0))
        *last_level = level;

      // reset retirement count so that next run succeeds
      retirementCountBuildNodes = 0; 
    }
  }
}


//////////////////////////////
//////////////////////////////
//////////////////////////////


KERNEL_DECLARE(cl_link_tree)(int n_nodes,
                            uint *n_children,
                            uint2 *node_bodies,
                            real4 *bodies_pos,
                            real4 corner,
                            uint2 *level_list,
                            uint* valid_list,
                            uint4 *node_keys,
                            uint4 *bodies_key,
                            uint  levelMin) {

  CUXTIMER("cl_link_tree");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes) return;

  uint2 bij  = node_bodies[id];
  uint level = (bij.x &  LEVELMASK) >> BITLEVELS;
  uint bi    =  bij.x & ILEVELMASK;
  uint bj    =  bij.y;

  real4 pos  = bodies_pos[bi];
  int4 crd;
  real domain_fac = corner.w;

  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif


  uint4 key = get_key(crd);


  /********* accumulate children *****/
  
  uint4 mask = get_mask(level - 1);
  key = make_uint4(key.x & mask.x, key.y & mask.y,  key.z & mask.z, 0); 

  uint2 cij;

  
  if(id > 0)
    cij = level_list[level-1];

  int ci;
  //JB, modified this since we don't use textures in find_key,
  //the function will fail because out of bound memory access when id==0
  if(id > 0)
    ci = find_key(key, cij, node_keys);
  else
    ci = 0;

  //ci now points to the node that is the parent, was used in previous group method
  //parent_id_list[id] = ci;

  mask = get_imask(mask);
  key = make_uint4(key.x | mask.x, key.y | mask.y, key.z | mask.z, 0);
  if (id > 0)   
    atomicAdd(&n_children[ci], (1 << 28));

  key = get_key(crd);
  mask = get_mask(level);
  key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

  /********* store the 1st child *****/

  cij = level_list[level+1];
  int cj = -1;

  cj = find_key(key, cij, node_keys);

  atomicOr(&n_children[id], cj); //Atomic since multiple threads can work on this

  uint valid =  id | (uint)(0 << 31); 

  
  if ((int)level > (int)(levelMin))
    if ((bj - bi) <= NLEAF)    
      valid = id | (uint)(1 << 31);   //Distinguish leaves and nodes

 valid_list[id] = valid; //If valid its a leaf otherwise a node
}

//Determines which level of node starts at which offset
KERNEL_DECLARE(gpu_build_level_list)(const int    n_nodes,
                                 const int    n_leafs,
                                       uint  *leafsIdxs,
                                       uint2 *node_bodies,
                                       uint  *valid_list)
{
  CUXTIMER("build_level_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes-n_leafs) return;

  const int nodeID = leafsIdxs[id+n_leafs];   //Get the idx into the node_bodies array

  int level_c, level_m, level_p;


  uint2 bij   = node_bodies[leafsIdxs[id+n_leafs]];    //current non-leaf
  level_c     = (bij.x &  LEVELMASK) >> BITLEVELS;

  if((id+1) < (n_nodes-n_leafs))        //The last node gets a default level
  {
    bij         = node_bodies[leafsIdxs[id+1+n_leafs]]; //next non-leaf
    level_p     = (bij.x &  LEVELMASK) >> BITLEVELS;
  }
  else
    level_p     = MAXLEVELS+5;  //Last is always an end

  //Compare level with the node before and node after
  if(nodeID == 0)
  {
    level_m = -1;    
  }
  else
  {
    bij         = node_bodies[ leafsIdxs[id-1+n_leafs]]; //Get info of previous non-leaf node
    level_m     =  (bij.x &  LEVELMASK) >> BITLEVELS;   
  }

  int valid0 = 0;
  int valid1 = 0;

  valid0 = (level_c != level_m) << 31 | (id+n_leafs);
  valid1 = (level_c != level_p) << 31 | (id+n_leafs);

  valid_list[id*2]   = valid0;
  valid_list[id*2+1] = valid1;

} //end build_level_list

#if 0
//Old version based on distance

//Finds nodes/leafs that will become groups
//After executions valid_list contains the 
//valid nodes/leafs that form groups
KERNEL_DECLARE(build_group_list2)(int    n_particles,
                                             uint  *validList,
                                             real4  *bodies_pos,
                                             const float DIST,
                                             int   *node_level_list,
                                             int   treeDepth)
{
  CUXTIMER("build_group_list2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;

  __shared__ int shmem[128];

  //Compact the node_level_list
  if(bid == 0)
  {
    if(threadIdx.x < (MAXLEVELS*2))
    {
      shmem[threadIdx.x] = node_level_list[threadIdx.x];
    }

    __syncthreads(); //Can most likely do without since its one warp

    //Only selection writes
    if(threadIdx.x < MAXLEVELS)
    {
      node_level_list[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
          node_level_list[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }//if bid == 0
  //end compact node level list

  //Note that we do not include the final particle
  //Since there is no reason to check it
  if (idx >= n_particles) return;

  //Get the current 
  float4 curPos, nexPos, prevPos;

  curPos  =  bodies_pos[idx];

  //Have to check the first and last to prevent out of bound access
  if(idx+1 == n_particles)
    nexPos  =  curPos;
  else
    nexPos = bodies_pos[idx+1];

  if(idx == 0)
    prevPos = curPos;
  else
    prevPos =  bodies_pos[idx-1];

  //Compute geometrical distance
  float dsPlus = ((curPos.x-nexPos.x)*(curPos.x-nexPos.x)) + 
                 ((curPos.y-nexPos.y)*(curPos.y-nexPos.y)) + 
                 ((curPos.z-nexPos.z)*(curPos.z-nexPos.z));

  float dsMin = ((curPos.x-prevPos.x)*(curPos.x-prevPos.x)) + 
                ((curPos.y-prevPos.y)*(curPos.y-prevPos.y)) + 
                ((curPos.z-prevPos.z)*(curPos.z-prevPos.z));

  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);

  //The extra possible split(s) if the distance between two particles is too large
  if(dsPlus > DIST) validEnd     = 1;
  if(dsMin  > DIST) validStart   = 1;
  
  //Last particle is always the end, n_particles dont have to be a multiple of NCRIT
  //so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid
  validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  validList[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}

#else
//New version based on top levels of the tree, uses top nodes/leafs which boundaries
//will become groups. After executions valid_list contains the valid nodes/leafs that form groups
KERNEL_DECLARE(build_group_list2)(const int   n_particles,
                                  uint       *validList,
                                  const uint2 startLevelBeginEnd,
                                  uint2      *node_bodies,
                                  int        *node_level_list,
                                  int         treeDepth)
{
  CUXTIMER("build_group_list2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;

  __shared__ int shmem[128];

  //Compact the node_level_list. From begin-end positions to just begin positions
  if(bid == 0)
  {
    if(threadIdx.x < (MAXLEVELS*2))
    {
      shmem[threadIdx.x] = node_level_list[threadIdx.x];
    }

    __syncthreads(); //Can most likely do without since its one warp if MAXLEVELS < 32

    //Only selection writes
    if(threadIdx.x < MAXLEVELS)
    {
      node_level_list[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
          node_level_list[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }//if bid == 0
  //end compact node level list

  //We do not include the final particle, since there is no reason to check it
  if (idx >= n_particles) return;

  //Now we get some info from tree-structure for the groups
  //Note that we do NOT include the last groups since it only sets
  //the final particle to invalid, which we will do by default anyway
  //this way we save a check on particle boundary

  //Use the end-indices of all tree-nodes above our minimum level
  if (idx < startLevelBeginEnd.y-1) //THe -1 to prevent last node
  {
    const uint2 bij          =  node_bodies[idx];
    const uint lastChild     =  bij.y;   

    //Set the boundaries, start and end 
    validList[2*lastChild - 1]  = (lastChild)   | (uint)(1 << 31);
    validList[2*lastChild]      = (lastChild)   | (uint)(1 << 31);
  }

  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);
  
  //Last particle is always the end, n_particles don't have
  //to be a multiple of NCRIT so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid, note only set it if we write something valid
  //otherwise we might overwrite the settings from the coarse group
  if(validStart) validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  if(validEnd)   validList[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}
#endif
 
//Store per particle the group id it belongs to
//and the start and end particle number of the groups  
KERNEL_DECLARE(store_group_list)(int    n_particles,
                                 int n_groups,
                                 uint  *validList,
                                 uint  *body2group_list,
                                 uint2 *group_list){
  CUXTIMER("store_group_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  //uint idx = bid * blockDim.x + tid;
  
  if(bid >= n_groups) return;

  int start = validList[2*bid];
  int end   = validList[2*bid+1];

  if((start + tid) < end)
  {
    body2group_list[start + tid] = bid;
  }

  if(tid == 0)
  {
     group_list[bid] = make_uint2(start,end);
  }
}


//////////// Functions specific for dust //////////////////

KERNEL_DECLARE(define_dust_groups)(int    n_particles,
					      real4  *dust_pos,
                                              uint  *validList)
{
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;


  //Note that we do not include the final particle
  //Since there is no reason to check it
  if (idx >= n_particles) return;

 
  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);


  //Get the current 
  float4 curPos, nexPos, prevPos;

  curPos  =  dust_pos[idx];

  //Have to check the first and last to prevent out of bound access
  if(idx+1 == n_particles)
    nexPos  =  curPos;
  else
    nexPos = dust_pos[idx+1];

  if(idx == 0)
    prevPos = curPos;
  else
    prevPos =  dust_pos[idx-1];

  //Compute geometrical distance
  float dsPlus = ((curPos.x-nexPos.x)*(curPos.x-nexPos.x)) + 
                 ((curPos.y-nexPos.y)*(curPos.y-nexPos.y)) + 
                 ((curPos.z-nexPos.z)*(curPos.z-nexPos.z));

  float dsMin = ((curPos.x-prevPos.x)*(curPos.x-prevPos.x)) + 
                ((curPos.y-prevPos.y)*(curPos.y-prevPos.y)) + 
                ((curPos.z-prevPos.z)*(curPos.z-prevPos.z));


  float DIST = 100;
  //The extra possible split(s) if the distance between two particles is too large
  if(dsPlus > DIST) validEnd     = 1;
  if(dsMin  > DIST) validStart   = 1;


  //Last particle is always the end, n_particles dont have to be a multiple of NCRIT
  //so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid
  if(validStart)
    validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  if(validEnd)
    validList[2*idx + 1] = (idx) | (uint)(validEnd   << 31);    
}

//JB: This one is slightly different from the store_group_list
//since  in my infinite wisdom I decided to make the comparisons
//slightly different when making the new define_dust_groups
KERNEL_DECLARE(store_dust_groups)(int    n_groups,
                                  uint  *validList,
                                  uint  *body2group_list,
                                  uint2 *group_list,
                                  uint  *activeDustGroups)
{
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
//   uint idx = bid * blockDim.x + tid;
  
  if(bid >= n_groups) return;

  int start = validList[2*bid];
  int end   = validList[2*bid+1];

  if((start + tid) <= end)
  {
     body2group_list[start + tid] = bid;
  }

  if(tid == 0)
  {
     group_list[bid] = (uint2) make_uint2(start,end+1);
     activeDustGroups[bid] = bid;
  }
}

//This function stores the predicted position and velocity
//in the original array. This is used since it reduces
//memory storage and memory reorders after sorting 
//It is slightly less accurate and therefore not used 
//for the real bodies. In the correct function we compute back
KERNEL_DECLARE(predict_dust_particles)(const int n_bodies,
                                                  float tc,
                                                  float tp,
                                                  real4 *pos,
                                                  real4 *vel,
                                                  real4 *acc,
                                                  uint  *body2grouplist,
                                                  uint  *valid_list){                                          
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint idx = bid * blockDim.x + tid;

  if (idx >= n_bodies) return;

  float4 p = pos [idx];
  float4 v = vel [idx];
  float4 a = acc [idx];

  float dt_cb  = tc - tp;

  p.x += v.x*dt_cb + a.x*dt_cb*dt_cb*0.5f;
  p.y += v.y*dt_cb + a.y*dt_cb*dt_cb*0.5f;
  p.z += v.z*dt_cb + a.z*dt_cb*dt_cb*0.5f;
  
  v.x += a.x*dt_cb;
  v.y += a.y*dt_cb;
  v.z += a.z*dt_cb;

  pos[idx] = p;
  vel[idx] = v;

  //This is needed to retain compatability with the original 
  //approximate gravity function
  int grpID = body2grouplist[idx];
  valid_list[grpID] = grpID; 
}

KERNEL_DECLARE(correct_dust_particles)(const int n_bodies,
                                                  float dt_cb,
                                                  uint   *active_list,
                                                  real4 *vel,
                                                  real4 *acc0,
                                                  real4 *acc1) {
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid =  threadIdx.y * blockDim.x + threadIdx.x;
  const int dim =  blockDim.x * blockDim.y;

  int idx = bid * dim + tid;
  if (idx >= n_bodies) return;

  //Check if particle is set to active during approx grav
  #ifdef DO_BLOCK_TIMESTEP
    if (active_list[idx] != 1) return;
  #endif

  float4 a0 = acc0[idx];
  float4 a1 = acc1[idx];
  float4  v = vel[idx];

  //Correct the velocity
  dt_cb *= 0.5f;
  v.x += (a1.x - a0.x)*dt_cb;
  v.y += (a1.y - a0.y)*dt_cb;
  v.z += (a1.z - a0.z)*dt_cb;

  //Store the corrected velocity, acceleration and the new time step info
  vel     [idx] = v;
  acc0    [idx] = a1;
}

/// End Dust Functions /////

//This is a simple place holder, example function
//depending on the data to be summarized
//it can be extended/modified
//Using atomics to prevent launch overhead when there
//are only few particles. Could be modified into 
//non atomic with Dynamic Parallelism
KERNEL_DECLARE(gpu_segmentedCoarseGroupBoundary)(
                                    const int n_coarse_groups, //Number of groups that have to be summarized
                                    const int n_groups,
                                    uint     *atomicValues,
                                    uint     *coarseGroupList,
                                    float4   *grpSizes,
                                    float4   *grpPositions,
                                    float4   *output_min,
                                    float4   *output_max)
{
//   const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  int *shmem2 = (int*)&shmem;

  uint bid;

  while(true)
  {
    //Get a ticket to the data-group that has to be processed
    if(threadIdx.x == 0)
    {
      bid         = atomicAdd(&atomicValues[0], 1);
      shmem2[0]    = bid;
    }
    __syncthreads();

    bid   = shmem2[0];

    if (bid >= n_coarse_groups) return; 

    const uint firstChild    =  coarseGroupList[bid];
    const uint lastChild     =  (bid == (n_coarse_groups-1)) ? n_groups : coarseGroupList[bid+1];
     
    __syncthreads();

#if 1
    float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
    float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

    volatile float3 *sh_rmin = (float3*)&shmem [ 0];
    volatile float3 *sh_rmax = (float3*)&shmem[256];
    sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
    sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

    // perform first level of reduction, reading from global memory, writing to shared memory
    const int blockSize   = blockDim.x;
    unsigned int i        = firstChild + tid;

    float4 pos;
    float4 size;

    while (i < lastChild)
    {
        if (i < lastChild)
        {
          pos             = grpPositions[i];
          size            = grpSizes[i];
          r_min.x = fminf(pos.x-size.x, r_min.x);
          r_min.y = fminf(pos.y-size.y, r_min.y);
          r_min.z = fminf(pos.z-size.z, r_min.z);
          r_max.x = fmaxf(pos.x+size.x, r_max.x);
          r_max.y = fmaxf(pos.y+size.y, r_max.y);
          r_max.z = fmaxf(pos.z+size.z, r_max.z);
        }
        if (i + blockSize < lastChild)
        {
          pos             = grpPositions[i+blockSize];
          size            = grpSizes[i+blockSize];
          r_min.x = fminf(pos.x-size.x, r_min.x);
          r_min.y = fminf(pos.y-size.y, r_min.y);
          r_min.z = fminf(pos.z-size.z, r_min.z);
          r_max.x = fmaxf(pos.x+size.x, r_max.x);
          r_max.y = fmaxf(pos.y+size.y, r_max.y);
          r_max.z = fmaxf(pos.z+size.z, r_max.z);
        }
        i += 2*blockSize;
      }

      sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
      sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

      __syncthreads();
      // do reduction in shared mem  
      if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
      if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
      if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

      if (tid < 32) 
      {
        sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
        sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
        sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
        sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
        sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
        sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
      }


      // write result for this block to global mem
      if (tid == 0)
      {
        //Compiler doesnt allow: volatile float3 = float3
    	output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y;
        output_min[bid].z = sh_rmin[0].z;
        output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y;
        output_max[bid].z = sh_rmax[0].z;
    }
#endif
  } //End while
}//end segmentedSummary


#if 0
//Function to mark the particles that are already assigned to a hash
KERNEL_DECLARE(build_parallel_grps)(
                             uint   compact_list_len,
                             uint   offset,
                             uint  *compact_list,
                             uint4 *bodies_key,
                             uint4 *parGrpBlockKey,
                             uint2 *parGrpBlockInfo,
                             uint  *startBoundary){

  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
//   uint id  = bid * blockDim.x + tid;

  if (bid >= compact_list_len) return;

  //Each block handles a bunch of particles
  uint  bi   = compact_list[bid*2];
  uint  bj   = compact_list[bid*2+1] + 1;

  #define NPARALLEL 1024

  if((bj - bi) > NPARALLEL)
  {
    if(tid == 0)
    {
      //Set the key to invalid and in item w a value that marks it invalid
      //This is redundent, and not used. Could be used for validation
      parGrpBlockInfo[offset+bid] =  (uint2){0, 0};
      parGrpBlockKey [offset+bid] = (uint4){0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0};
    }
    return;
  }

  int nparticles = bj-bi;

  //Only has to be done by one thread, other threads are just here for data writing

  //For each block we store the first particle, the particles key and the last particle
  //key is 3 uints -> so we have the last one free keep free
  //since we cannot store the pid and number of particles
  //if N-particles is high (> 2M)
  if(tid == 0)
  {
    uint4 key  = bodies_key[bi];
    key.w = bj-bi;

    uint2 blockInfo = (uint2){bi, bj};

    parGrpBlockInfo[offset+bid] = blockInfo;
    parGrpBlockKey [offset+bid] = key;


    //Set the start boundary, which indicates the particle
    //that forms the start of the parallel hash and refers
    //to the group that has this particle as start
    startBoundary[bi] = (uint)( offset+bid | (uint)(1 << 31));
  } //tid == 0


  for(int i=0; i < nparticles; i += blockDim.x)
  {
    if(i + tid < nparticles)
    {
      //sets the key to FF to indicate the body is used
      bodies_key[bi+i+tid] = (uint4){0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF};
    }
  } //for nparticles

} //end cl_build_parallel_reduce

#else

//Function to mark the particles that are already assigned to a hash
KERNEL_DECLARE(gpu_build_parallel_grps)(
                             uint   compact_list_len,
                             uint   offset,
                             const uint NPARALLEL,
                             uint  *compact_list,
                             uint4 *bodies_key,
                             uint4 *parGrpBlockKey,
                             uint2 *parGrpBlockInfo,
                             uint  *startBoundary){

  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
//   uint id  = bid * blockDim.x + tid;

  if (bid >= compact_list_len) return;

  //Each block handles a bunch of particles
  uint  bi   = compact_list[bid*2];
  uint  bj   = compact_list[bid*2+1] + 1;

//  #define NPARALLEL 1024

  if((bj - bi) > NPARALLEL)
  {
    if(tid == 0)
    {
      //Set the key to invalid and in item w a value that marks it invalid
      //the .w item is redundant, and not used. Could be used for validation
      parGrpBlockInfo[offset+bid] = make_uint2(0, 0);
      parGrpBlockKey [offset+bid] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0);
    }
    return;
  }

  int nparticles = bj-bi;

  //Only has to be done by one thread, other threads are just here for data writing

  //For each block we store the first particle, the particles key and the last particle
  //key is 3 uints -> so we have the last one free keep free
  //since we cannot store the pid and number of particles
  //if N-particles is high (> 2M)
  if(tid == 0)
  {
    uint4 key  = bodies_key[bi];
    key.w = bj-bi;

    uint2 blockInfo = make_uint2(bi, bj);

    parGrpBlockInfo[offset+bid] = blockInfo;
    parGrpBlockKey [offset+bid] = key;


    //Set the start boundary, which indicates the particle
    //that forms the start of the parallel hash and refers
    //to the group that has this particle as start
    startBoundary[bi] = (uint)( offset+bid | (uint)(1 << 31));
  } //tid == 0


  for(int i=0; i < nparticles; i += blockDim.x)
  {
    if(i + tid < nparticles)
    {
      //sets the key to FF to indicate the body is used
      bodies_key[bi+i+tid] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF);
    }
  } //for nparticles

} //end cl_build_parallel_reduce
#endif

//static __device__ uint retirementCountSegmentedSummaryBasic = 0;
//This is a simple place holder, example function
//depending on the data to be summarized
//it can be extended/modified
//Using atomics to prevent launch overhead when there
//are only few particles. Could be modified into
//non atomic with Dynamic Parallism
KERNEL_DECLARE(gpu_segmentedSummaryBasic) (
                                            const int n_groups, //Number of groups that have to be summarized
                                            uint     *validGroups,
                                            uint     *atomicValues,
                                            uint2    *hashGroupInfo,    //parGrpBlockInfo
                                            uint4    *hashGroupKey,     //parGrpBlockKey
                                            uint4    *hashGroupResult,  //parallelHashes
                                            uint4    *sourceData)       //bodies_key
{
//   const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;


  volatile __shared__ int shmem[256];

  uint bid;

  while(true)
  {
    //Get a ticket to the data-group that has to be processed
    if(threadIdx.x == 0)
    {
      bid         = atomicAdd(&atomicValues[0], 1);
      shmem[0]    = bid;
    }
    __syncthreads();


    bid   = shmem[0];

    if (bid >= n_groups) return;

    int hashGrpID        = validGroups[bid];

    __syncthreads();


#if 0  //We can use the below when we add interactions count
// we reduce multiple elements per thread.  The number is determined by the
// number of active thread blocks (via gridSize).  More blocks will result
// in a larger gridSize and therefore fewer elements per thread

    //Start and end index of the data values to be processed
    int start = hashGroupInfo[hashGrpID].x;
    int end   = hashGroupInfo[hashGrpID].y;


//based on reduce6 example
    volatile int *sh_sum = (int*)&shmem [ 0];

    int localSum = 0;
    sh_sum[tid]  = 0;

    // perform first level of reduction, reading from global memory, writing to shared memory
    const int blockSize   = blockDim.x;
    unsigned int i        = start;


    while (i < end) {
      if(i + tid < end)
      {
        //Ad a reduction over the interactions here later on
        localSum += 1;
      }
      i += blockSize;
    }

    sh_sum[tid] = localSum;

    __syncthreads();
    // do reduction in shared mem
    if(blockDim.x >= 512) if (tid < 256) { sh_sum[tid] = localSum = localSum + sh_sum[tid + 256];} __syncthreads();
    if(blockDim.x >= 256) if (tid < 128) { sh_sum[tid] = localSum = localSum + sh_sum[tid + 128];} __syncthreads();
    if(blockDim.x >= 128) if (tid < 64)  { sh_sum[tid] = localSum = localSum + sh_sum[tid + 64];} __syncthreads();

    if (tid < 32)
    {
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 32];
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 16];
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 8];
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 4];
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 2];
      sh_sum[tid] = localSum = localSum + sh_sum[tid + 1];
    }
    #endif

    // write result for this block to global mem
    if (tid == 0)
    {
      uint4 data           = hashGroupKey[hashGrpID];
//       data.w               = sh_sum[0];
      hashGroupResult[bid] = data;
    }

  } //End while
}//end segmentedSummary




